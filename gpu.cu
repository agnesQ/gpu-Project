#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "common_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "lodepng.h"
#include "lodepng.cpp"

using namespace std;

int NUM_BLOCKS;
int BLOCK_SIZE;

vector<unsigned char> image; 
vector<vector<unsigned char> > img;
vector<vector<unsigned char> > original_img;
unsigned char *img1;
unsigned long long *rowChecksum1;
unsigned long long *columnChecksum1;
unsigned char *OrigintotalRowSum;
unsigned char *OrigintotalColumnSum;
unsigned char *ScrambledRowSum;
unsigned char *ScrambledColSum;
unsigned width, height;
unsigned size;
unsigned num;

hipError_t gpuchecksum(unsigned char *imge, unsigned long long *rowCS, unsigned long long *colCS, unsigned char *OriRowCS, unsigned char *OriColCS, unsigned char *ScrRowCS, unsigned char *ScrColCS, float* Runtimes);

__global__ void CheckSumKernelG(unsigned char *Gimg, unsigned long long *GrowCS, unsigned long long *GcolCS, unsigned char *GOriRowCS, unsigned char *GOriColCS, unsigned char *GScrRowCS, unsigned char *GScrColCS, unsigned height, unsigned width, unsigned size, unsigned num) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	int i, j, k, s;
	unsigned long long t, tmp;
	unsigned char count;
	int flag;
	
	flag = (gid/height)%height; 
	
	switch(flag){
	case 0:
			{i = gid;
			j = i/size;
			//s = 8 *(i%size);
			s = 8 *(size - 1 - i%size);
			t = (unsigned long long)255 << s;
			count = 0;
			for (k=0; k<num; ++k) {
				tmp = GcolCS[j*num+k] & t;
				count ^= (tmp >> s);
			}
			GOriRowCS[i] = count;
			break;}
	case 1:
			{i = gid - height;
			j = i/size;
			s = 8 *(size -1 - i%size);
			//s = 8 *(i%size);
			t = (unsigned long long)255 << s;
			count = 0;
			for (k = 0; k < num; ++k) {
				tmp = GrowCS[k*num +j] & t;
				count ^= (tmp >> s);
			}
			GOriColCS[i] = count;
			break;}

	case 2:
			{i = gid - (height+width);
			count = 0;
			for (j = 0; j < width; j++) {
				count ^= Gimg[i*width+j];
			}
			GScrRowCS[i] = count;
			break;}
			
	case 3:
			{i = gid - (2*height+width);
			count = 0;
			for (j = 0; j < height; j++) {
				count ^= Gimg[j*width+i];
			}
			GScrColCS[i] = count;
			break;}
	}

}

void Read_in(const char* filename)
{
  
  unsigned error = lodepng::decode(image, width, height, filename);
  if(error) cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;
  
  img1 = (unsigned char*)malloc(height*width*sizeof(unsigned char));

  for (int i = 0; i < height; ++i)
  {
  	vector<unsigned char> tmp;
  	long k = 4 * i * width;
  	for (int j = 0; j < 4 * width; j += 4)
  	{
  		tmp.push_back(image[k + j]);
  		img1[i*width+j/4] = image[k + j];
  	}
  	img.push_back(tmp);
  }
}



int main(int argc, char** argv)
{
	if (argc < 5)
	{
		cout << "input error" << endl;
		return 1;
	}
	Read_in(argv[1]);
	long ExeTime;
  	struct timeval t;
  	double start, end;
  	float GPURuntimes[4]; 
  	hipError_t cudaStatus;

	
	size = argv[3][0] - '0';
	num = width / size;
	BLOCK_SIZE = atoi(argv[4]);
	if (2*(height+width)%BLOCK_SIZE==0) NUM_BLOCKS = 2*(height+width)/BLOCK_SIZE;
	else NUM_BLOCKS = 2*(height+width)/BLOCK_SIZE+1;
	int count = 0;
	int res = 0;
	vector<int> row(height, -1);
	vector<int> column(width, -1);
	
	rowChecksum1 = (unsigned long long*)malloc(num*num*sizeof(unsigned long long));
	columnChecksum1 = (unsigned long long*)malloc(num*num*sizeof(unsigned long long));

	int p = 0;
	ifstream fin;
	fin.open(argv[2]);
	unsigned long long i,j;
	char c;
	if (fin.is_open()) {
	  while (!fin.eof()) {
	    fin >> i >> c >> j;
	    rowChecksum1[p] = i;
	    columnChecksum1[p] = j;
	    p++;
	 }
	}	
	fin.close();

	OrigintotalRowSum = (unsigned char*)malloc(height*sizeof(unsigned char));
	ScrambledRowSum = (unsigned char*)malloc(height*sizeof(unsigned char));
	OrigintotalColumnSum = (unsigned char*)malloc(width*sizeof(unsigned char));
	ScrambledColSum = (unsigned char*)malloc(width*sizeof(unsigned char));

	cudaStatus = gpuchecksum(img1, rowChecksum1, columnChecksum1, OrigintotalRowSum, OrigintotalColumnSum, ScrambledRowSum, ScrambledColSum, GPURuntimes);
 	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n gpuchecksum failed!");
		return 1;
	}

	printf("\nKERNEL = ElimtKernelG ...\n");
	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n",GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
	printf("-----------------------------------------------------------------\n");

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	
	
	for (int i = 0; i < height; ++i)
	{
		count = 0;
		res = 0;
		for (int j = 0; j < height; ++j)
		{
			if (OrigintotalRowSum[i] == ScrambledRowSum[j])
			{
				count++;
				res = j;
			}
		}
		row[i] = res;

	}

	for (int i = 0; i < width; ++i)
	{
		count = 0;
		res = 0;
		for (int j = 0; j < width; j++)
		{
			if (OrigintotalColumnSum[i] == ScrambledColSum[j])
			{
				count++;
				res = j;
			}
		}
		column[i] = res;

	}

	original_img = img;
	for (int i = 0; i < height; ++i)
	{
		for (int j = 0; j < width; ++j)
		{
			original_img[i][j] = img[row[i]][column[j]];
		}
	}
	
	//encodeOneStep("original.png", width, height);

	FILE *fp;
	fp=fopen("recovered_result.txt", "w");
	for(int i=0; i<height; i++)
	{
		for(int j=0; j<width; j++)
		{
			fprintf(fp, "%d",original_img[i][j]);
			if(j!=width)
				fprintf(fp, ",");
		}
		fprintf(fp, "\n");
	}
	fclose(fp);


	free(img1);
	free(rowChecksum1);
	free(columnChecksum1);
	free(OrigintotalRowSum);
	free(OrigintotalColumnSum);
	free(ScrambledRowSum);
	free(ScrambledColSum);

	return 0;
}

hipError_t gpuchecksum(unsigned char *imge, unsigned long long *rowCS, unsigned long long *colCS, unsigned char *OriRowCS, unsigned char *OriColCS, unsigned char *ScrRowCS, unsigned char *ScrColCS, float* Runtimes) {
	hipEvent_t time1, time2, time3, time4;
  	
  	unsigned char *Gimg;
  	unsigned long long *GrowCS, *GcolCS;
  	unsigned char *GOriRowCS, *GOriColCS, *GScrRowCS, *GScrColCS;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
  	cudaStatus = hipSetDevice(0);
  	if (cudaStatus != hipSuccess) {
     fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	   goto Error;
  	}
 	hipEventCreate(&time1);
  	hipEventCreate(&time2);
  	hipEventCreate(&time3);
  	hipEventCreate(&time4);

	hipEventRecord(time1, 0);
  // Allocate GPU buffer for inputs and outputs (sortenuse)

	cudaStatus = hipMalloc((void**)&Gimg, height*width*sizeof(unsigned char));
  if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMalloc failed!");
         goto Error;
  }

	cudaStatus = hipMalloc((void**)&GrowCS, num*num*sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GcolCS, num*num*sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GOriRowCS, height*sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GOriColCS, width*sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GScrRowCS, height*sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&GScrColCS, width*sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(Gimg, imge, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(GrowCS, rowCS, num*num*sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(GcolCS, colCS, num*num*sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipEventRecord(time2, 0);

	CheckSumKernelG<<<NUM_BLOCKS, BLOCK_SIZE>>>(Gimg, GrowCS, GcolCS, GOriRowCS, GOriColCS, GScrRowCS, GScrColCS, height, width, size, num);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}


	hipEventRecord(time3, 0);
	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(OriRowCS, GOriRowCS, height*sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(OriColCS, GOriColCS, width*sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ScrRowCS, GScrRowCS, height*sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ScrColCS, GScrColCS, width*sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipEventRecord(time4, 0);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

	Error:
	hipFree(Gimg);
	hipFree(GrowCS);
	hipFree(GcolCS);
	hipFree(GOriRowCS);
	hipFree(GOriColCS);
	hipFree(GScrRowCS);
	hipFree(GScrColCS);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);

	return cudaStatus;
}